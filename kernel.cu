#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "kernel.h"
// #include "kernel.cu"
// #include "dev_array.h"
#include <math.h>

using namespace std;

__global__ void matrixMultiplicationKernel(double* A, double* B, double* C, int N) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    C[ROW * N + COL] = tmpSum;
}


extern "C" void multiplication_GPU(double *l1, double *l2, double **conn, int n1, int n2);
void multiplication_GPU(double *l1, double *l2, double **conn, int n1, int n2)
{

	int N = 1024;
    int SIZE = N*N;

	int input = n1;
	int hiddenlayer = n2;
	
	int inputSize = input * sizeof(double);
	int hiddenSize = hiddenlayer * sizeof(double);

	int connection = input * hiddenlayer;
	int connectionSize = connection * sizeof(double);

	double h_3[input*hiddenSize];

	for (int i = 0; i < input; i++)
			for(int j = 0; j < hiddenSize; j++)
					h_3[i*hiddenSize + j] = conn[i][j];


	double *d_input;
	double *d_conn;
	double *d_pp;
	double *d_out;

	hipMalloc((void **)&d_input, inputSize);
	hipMalloc((void **)&d_conn, connectionSize);
	hipMalloc((void **)&d_pp, connectionSize);
	hipMalloc((void **)&d_out, hiddenSize);

	hipMemcpy(d_input, l1, inputSize, hipMemcpyHostToDevice);
	hipMemcpy(d_conn, h_3, connectionSize, hipMemcpyHostToDevice);

	matrixMultiplicationKernel<<<hiddenlayer,inputSize>>>(d_pp, d_input, d_conn, hiddenlayer);


	hipMemcpy(l2, d_out, hiddenSize, hipMemcpyDeviceToHost);
	hipFree(d_conn);
	hipFree(d_input);
	hipFree(d_pp);
	hipFree(d_out);

}